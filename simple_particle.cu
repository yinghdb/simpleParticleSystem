#include "hip/hip_runtime.h"
#include "simple_particle.cuh"
#include ""
#include "hip/device_functions.h"
#include "math_functions.h"
#include <stdio.h>

__constant__ simpleParticleSystem d_sps[1];

__global__ void generateParticles();

__global__ void renderParticles(uchar4* devPtr, int img_width, int img_height);

__global__ void updateParticles(float passed_time);

__device__ float2 get_normal_vector(float rand_num);

__device__ float get_energy(float2 p1, float2 p2, float dist_bound_powerd);

__device__ uchar4 get_color_from_energy(float energy);

__device__ float2 get_acceleration(int index);

__device__ void update_particle_velocity(int index, float2 acc);

__device__ int update_particle_possition(int index); //return whether the particle is dead

void init_particles_cuda(simpleParticleSystem &sps) {
	int max_num_particles = sps.MAX_PARTICLE_SIZE;
	int one_batch_num_particles = sps.ONE_BATCH_PARTICLE_SIZE;

	hipMalloc((void**)&sps.energy, sizeof(*sps.energy)*max_num_particles);
	hipMalloc((void**)&sps.position, sizeof(*sps.position)*max_num_particles);
	hipMalloc((void**)&sps.velocity, sizeof(*sps.velocity)*max_num_particles);
	hipMalloc((void**)&sps.remain_time, sizeof(*sps.remain_time)*max_num_particles);
	hipMalloc((void**)&sps.rand_data, sizeof(*sps.rand_data)*one_batch_num_particles*3);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Memory Allocation Error: %s\n", hipGetErrorString(err));
}

void destroy_particles_cuda(simpleParticleSystem &sps) {
	hipError_t er;

	er = hipFree(sps.energy);
	er = hipFree(sps.position);
	er = hipFree(sps.velocity);
	er = hipFree(sps.remain_time);
	er = hipFree(sps.rand_data);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Memory Free Error: %s\n", hipGetErrorString(err));
}

void copy_to_device_sps(simpleParticleSystem &sps) {
	hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_sps), &sps, sizeof(simpleParticleSystem));

	if (err != hipSuccess)
		printf("Constant Memory Copy Error: %s\n", hipGetErrorString(err));
}

void generate_particles(int thread_size) {
	generateParticles << < 1, thread_size >> > ();
	//generateParticlesLine <<< 1, sps.ONE_BATCH_PARTICLE_SIZE >>> (
	//	sps.position, sps.velocity_orientation, sps.velocity, sps.remain_time, sps.rand_data, sps.ONE_BATCH_PARTICLE_SIZE,
	//	sps.MAX_PARTICLE_SIZE, sps.generator_line[0], sps.generator_line[1], sps.MAX_VELOCITY, sps.MIN_VELOCITY, sps.LIFE_TIME
	//);
}

void render_particles(uchar4* devPtr, int img_width, int img_height) {
	int thread_dim = 16;
	int grid_dim_x = (img_width + thread_dim - 1) / thread_dim;
	int grid_dim_y = (img_height + thread_dim - 1) / thread_dim;
	dim3 grids(grid_dim_x, grid_dim_y);
	dim3 threads(thread_dim, thread_dim);
	renderParticles << <grids, threads >> > (devPtr, img_width, img_height);
}

__global__ void generateParticles()
{
	float2 *position = (*d_sps).position;
	float2 *velocity = (*d_sps).velocity;
	float *remain_time = (*d_sps).remain_time;
	float *rand = (*d_sps).rand_data;
	int generate_size = (*d_sps).ONE_BATCH_PARTICLE_SIZE;
	int max_size = (*d_sps).MAX_PARTICLE_SIZE;

	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	__shared__ unsigned int generate_start_index;

	//get the particle generate block pos
	if (index == 0) {
		max_size -= generate_size;
		generate_start_index = 0;
		while (generate_start_index <= max_size) {
			if (remain_time[generate_start_index] == 0)
				break;
			generate_start_index += generate_size;
		}
	}

	__syncthreads();

	if (generate_start_index > max_size)
		return;

	int pid = generate_start_index + index; 
	float x;
	float y;
	float2 velocity_orientation;
	float n_velocity;

	//generate rand position and velocity
	switch ((*d_sps).TYPE)
	{
	case LineGenerator:
		x = rand[index] * ((*d_sps).generator_line[0].x - (*d_sps).generator_line[1].x) + (*d_sps).generator_line[1].x;
		y = rand[index] * ((*d_sps).generator_line[0].y - (*d_sps).generator_line[1].y) + (*d_sps).generator_line[1].y;
		position[pid] = make_float2(x, y);

		rand += generate_size;
		pid = generate_start_index + index;
		velocity_orientation = get_normal_vector(rand[index]);

		rand += generate_size;
		n_velocity = rand[index] * ((*d_sps).MAX_VELOCITY - (*d_sps).MIN_VELOCITY) + (*d_sps).MIN_VELOCITY;
		velocity[pid].x = n_velocity * velocity_orientation.x;
		velocity[pid].y = n_velocity * velocity_orientation.y;
		break;
	case CircleGenerator:
		float rand_pos = rand[index];
		float2 vec = get_normal_vector(rand_pos);
		x = vec.x * (*d_sps).generator_radius.x + (*d_sps).generator_center.x;
		y = vec.y * (*d_sps).generator_radius.y + (*d_sps).generator_center.y;
		position[pid] = make_float2(x, y);

		rand += generate_size;
		pid = generate_start_index + index;
		float rand_orient = rand[index];
		rand_orient = rand_pos + (rand_orient / 2 - rand_orient / 4);
		velocity_orientation = get_normal_vector(rand_orient);

		rand += generate_size;
		n_velocity = rand[index] * ((*d_sps).MAX_VELOCITY - (*d_sps).MIN_VELOCITY) + (*d_sps).MIN_VELOCITY;
		velocity[pid].x = n_velocity * velocity_orientation.x;
		velocity[pid].y = n_velocity * velocity_orientation.y;
		break;
	default:
		break;
	}

	//generate remain time
	remain_time[pid] = (*d_sps).LIFE_TIME;
}

__global__ void renderParticles(uchar4* devPtr, int img_width, int img_height) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= img_width || y >= img_height)
		return;

	
	if (!(x >= (*d_sps).BOUND_BOX[0] && x <= (*d_sps).BOUND_BOX[2]
		&& y <= (*d_sps).BOUND_BOX[1] && y >= (*d_sps).BOUND_BOX[3]))
		return;

	int generate_size = (*d_sps).ONE_BATCH_PARTICLE_SIZE;
	int max_size = (*d_sps).MAX_PARTICLE_SIZE;
	float energy = 0;
	float dist_bound_powerd = (*d_sps).ENERGY_SCOPE * (*d_sps).ENERGY_SCOPE;
	float2 pos = make_float2(x, y);
	for (int start_index = 0; start_index < max_size - generate_size; start_index += generate_size)
	{
		if ((*d_sps).remain_time[start_index] == 0)
			continue;
		//here we do not render the first particle of the batch
		for (int index = start_index + 1; index < start_index + generate_size; ++index) {
			if ((*d_sps).remain_time[index] != 0) {
				energy += get_energy((*d_sps).position[index], pos, dist_bound_powerd);
				if (energy >= 1) {
					energy = 1;
					break;
				}
			}
		}
		if (energy >= 1) {
			break;
		}
	}


	int offset = x + y * img_width;
	devPtr[offset] = get_color_from_energy(energy);
}


__global__ void updateParticles(float passed_time) {
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int strip = gridDim.x * blockDim.x;
	unsigned int start_index = blockIdx.x*blockDim.x;

	__shared__ int living_particle_num;

	while (index < (*d_sps).MAX_PARTICLE_SIZE) {
		living_particle_num = 0;
		__syncthreads();

		if ((*d_sps).remain_time[start_index] == 0)
			continue;
		
		float2 acc = get_acceleration(index);

		index += strip;
		start_index += strip;
	}
}


__device__ float2 get_normal_vector(float rand_num) {
	float x, y;
	sincosf(rand_num*2*PI, &y, &x);

	return make_float2(x, y);
}

__device__ float get_energy(float2 p1, float2 p2, float dist_bound_powerd) {
	float dx = p1.x - p2.x;
	float dy = p1.y - p2.y;
	float dist_powered = dx*dx + dy*dy;

	if (dist_powered > dist_bound_powerd)
		return 0;
	if (dist_powered == 0)
		return 0.5;
	return 0.5 / dist_powered;
}

__device__ uchar4 get_color_from_energy(float energy) {
	unsigned char r = 255 * energy;
	unsigned char g = 180 * energy;
	unsigned char b = 60 * energy;
	unsigned char w = 255 * energy;

	return make_uchar4(r, g, b, w);
}

__device__ float2 get_acceleration(int index) {
	return make_float2(20.0, 0);
}

__device__ void update_particle_velocity(int index, float2 acc, float passed_time) {
	(*d_sps).velocity[index].x += acc.x * passed_time;
	(*d_sps).velocity[index].y += acc.y * passed_time;
}

__device__ int update_particle_position(int index, float passed_time) {
	if ((*d_sps).remain_time[index] - passed_time <= 0) {
		(*d_sps).remain_time[index] = 0;
		return 0;
	}
	
	float2 *pos = &(*d_sps).position[index];
	(*pos).x += (*d_sps).velocity[index].x * passed_time;
	(*pos).y += (*d_sps).velocity[index].y * passed_time;

	if ((*pos).x > (*d_sps).LIFE_BOUND[0] && (*pos).x < (*d_sps).LIFE_BOUND[2]
		&& (*pos).y < (*d_sps).LIFE_BOUND[1] && (*pos).y < (*d_sps).LIFE_BOUND[3]) {
		return 1;
	}

	(*d_sps).remain_time[index] = 0;
	return 0;
}